#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include "a3.hpp"

// define a block size
const int blockSize = 256;

// gaussian function
__device__ float gaussian(float x) {
    return expf(-0.5 * x * x) / (sqrtf(2.0 * M_PI));
}

__global__ void gaussian_kde_kernel(const float* x, float* y, int n, float h) {
    // shared memory for data
    __shared__ float sdata[blockSize];
    // calculate the global thread index
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    // check if the thread index is within the bounds of the input data
    if (gidx < n) {
        float sum = 0.0;

        // Iterate over the input data
        for (int i = 0; i < n; i += blockDim.x) {

            // load data into shared memory
            if (i + threadIdx.x < n) {
                sdata[threadIdx.x] = x[i + threadIdx.x];
            } else {
                sdata[threadIdx.x] = 0;
            }
            __syncthreads();

            float x_i = x[gidx];

            // compute sum
            for (int j = 0; j < blockDim.x; ++j) {
                if (i + j < n) {
                    float diff = (x_i - sdata[j]) / h;
                    sum += gaussian(diff);
                }
            }

            __syncthreads();
        }
    
        // update y in global memory
        y[gidx] = sum / (n * h);
    }
}

void gaussian_kde(int n, float h, const std::vector<float>& x, std::vector<float>& y) {
    float *d_x, *d_y;
    
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));

    hipMemcpy(d_x, x.data(), n * sizeof(float), hipMemcpyHostToDevice);

    int gridSize = (n + blockSize - 1) / blockSize;

    gaussian_kde_kernel<<<gridSize, blockSize>>>(d_x, d_y, n, h);

    hipMemcpy(y.data(), d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}
